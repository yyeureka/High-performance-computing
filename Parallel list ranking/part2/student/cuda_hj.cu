#include "hip/hip_runtime.h"
#include <unistd.h>
#include "listutils.h"

#define TRUE  1
#define FALSE 0

__device__ int isSubHead(long node, long head, long s) {
	if (NIL == node) {
		return FALSE;
	}
	if ((head == node) || ((head >= s) && (node < (s - 1))) 
	|| ((head < s) && (node < s))) {
		return TRUE;
	}
	return FALSE;
}

__global__ void link(long head, long* next, long* sub_head, long* next_head, long* head_rank, size_t s) {
	long r;
	long node;

	size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < s) {
		node = sub_head[i];
		r = 0;

		while (NIL != node) {
			r++;
			node = next[node];

			if (TRUE == isSubHead(node, head, s)) {
				if ((head >= s) && ((s - 1) == i)) {
					next_head[s - 1] = node;
				}
				else {
					next_head[sub_head[i]] = node;
				}
				head_rank[node] = r;
				break;
			}
		}
	}
}

__global__ void rank_sublist(long head, long* next, long* rank, long* sub_head, size_t s) {
	long r;
	long node;

	size_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < s) {
		node = sub_head[i];
		r = rank[node];

		while (NIL != node) {
			rank[node] = r++;
			node = next[node];

			if (TRUE == isSubHead(node, head, s)) {
				break;
			}
		}
	}
}

extern "C" void parallelListRanks(const long head, const long* next, long* rank, const size_t n)
{
	/* Your Code Goes Here */
	size_t s = n / 50;
	long sub_head[s], next_head[s], head_rank[s];
	long r;
	long node;

	dim3 block_config(32);
	dim3 grid_config(s - 1 / block_config.x);
	long* d_next;
	long* d_rank;
	long* d_sh;
	long* d_nh;
	long* d_hr;
	hipMalloc((void**)&d_next, sizeof(long) * n);
	hipMalloc((void**)&d_rank, sizeof(long) * n);
	hipMalloc((void**)&d_sh, sizeof(long) * s);
	hipMalloc((void**)&d_nh, sizeof(long) * s);
	hipMalloc((void**)&d_hr, sizeof(long) * s);

	// Initialize
	for (size_t i = 0; i < s; i++) {
		sub_head[i] = i;
		next_head[i] = NIL;
	}
	if (head >= s) {
		sub_head[s - 1] = head;
	}
	rank[head] = 0;

	// Link sub-lists
	hipMemcpy(d_next, next, sizeof(long) * n, hipMemcpyHostToDevice);
	hipMemcpy(d_sh, sub_head, sizeof(long) * s, hipMemcpyHostToDevice);
	hipMemcpy(d_nh, next_head, sizeof(long) * s, hipMemcpyHostToDevice);
	hipMemcpy(d_hr, head_rank, sizeof(long) * s, hipMemcpyHostToDevice);
	link<<<grid_config, block_config>>>(head, d_next, d_sh, d_nh, d_hr, s);
	hipMemcpy(next_head, d_nh, sizeof(long) * s, hipMemcpyDeviceToHost);
	hipMemcpy(head_rank, d_hr, sizeof(long) * s, hipMemcpyDeviceToHost);

	// Rank sub-heads
	node = head;
	r = 0;
	while (NIL != node) {
		if (head == node) {
			if (head >= s) {
				node = next_head[s - 1];
			}
			else {
				node = next_head[node];
			}
		}
		else {
			rank[node] = head_rank[node] + r;
			r = rank[node];
			node = next_head[node];
		}
	}

	// Rank sub-lists
	hipMemcpy(d_rank, rank, sizeof(long) * n, hipMemcpyHostToDevice);
	hipMemcpy(d_next, next, sizeof(long) * n, hipMemcpyHostToDevice);
	hipMemcpy(d_sh, sub_head, sizeof(long) * s, hipMemcpyHostToDevice);
	rank_sublist<<<grid_config, block_config>>>(head, d_next, d_rank, d_sh, s);
	hipMemcpy(rank, d_rank, sizeof(long) * n, hipMemcpyDeviceToHost);

	hipFree(d_next);
	hipFree(d_rank);
	hipFree(d_sh);
	hipFree(d_nh);
	hipFree(d_hr);
}
